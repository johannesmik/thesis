#include "hip/hip_runtime.h"
// Author: Johannes Mikulasch
// July 2015

#ifndef INTENSITIES_CU
#define INTENSITIES_CU

#include "lights.cu"


__device__ float intensity(const float3 &normal, const float3 &w) {

  if (w.z == 0)
     return 0;

  const float ambient = 0.2;
  const float albedo = 0.8;
  const float falloff = 0.0;
  const float3 camera = make_float3(0, 0, 0);
  const float3 light = light_point(w);

  return clamp(ambient + attenuation(falloff, dist(w, camera)) * albedo * dot(normal, light), 0, 1);
}

#endif
